#include "hip/hip_runtime.h"
/** Detray library, part of the ACTS project (R&D line)
 *
 * (c) 2023 CERN for the benefit of the ACTS project
 *
 * Mozilla Public License Version 2.0
 */

#include "detray/definitions/cuda_definitions.hpp"
#include "propagation.hpp"

namespace detray {

// Propagation configurations
inline constexpr detray::scalar path_limit{2.f * unit<scalar>::m};

/// Kernel that runs the entire propagation loop
__global__ void propagation_example_kernel(
    typename detray::detector_host_t::detector_view_type det_data,
    const vecmem::data::vector_view<
        detray::free_track_parameters<example::transform3>>
        tracks_data,
    vecmem::data::jagged_vector_view<detray::intersection_t> candidates_data) {

    int gid = threadIdx.x + blockIdx.x * blockDim.x;

    // Setup device-side track collection
    vecmem::device_vector<detray::free_track_parameters<example::transform3>>
        tracks(tracks_data);

    if (gid >= tracks.size()) {
        return;
    }

    // Setup of the device-side detector
    detray::detector_device_t det(det_data);
    // Setup of the avigator cache
    vecmem::jagged_device_vector<detray::intersection_t> candidates(
        candidates_data);
    // Setup of the device b-field
    detray::detector_device_t::bfield_type B_field = det.get_bfield();

    // Create propagator from a stepper and a navigator
    detray::propagator_t p(detray::stepper_t{}, detray::navigator_t{});

    // Create actor states
    detray::pathlimit_aborter::state aborter_state{path_limit};
    detray::parameter_transporter<example::transform3>::state
        transporter_state{};
    detray::pointwise_material_interactor<example::transform3>::state
        interactor_state{};
    detray::parameter_resetter<example::transform3>::state resetter_state{};

    auto actor_states = ::detray::tie(aborter_state, transporter_state,
                                      interactor_state, resetter_state);

    // Create the propagator state for the track
    detray::propagator_t::state state(tracks[gid], B_field, det,
                                      candidates.at(gid));

    // Run propagation
    p.propagate(state, actor_states);
}

void propagation_example(
    typename detray::detector_host_t::detector_view_type det_data,
    const vecmem::data::vector_view<
        detray::free_track_parameters<example::transform3>>
        tracks_data,
    vecmem::data::jagged_vector_view<detray::intersection_t> candidates_data) {

    int thread_dim = 2 * WARP_SIZE;
    int block_dim = tracks_data.size() / thread_dim + 1;

    // run the example kernel
    propagation_example_kernel<<<block_dim, thread_dim>>>(det_data, tracks_data,
                                                          candidates_data);

    // cuda error check
    DETRAY_CUDA_ERROR_CHECK(hipGetLastError());
    DETRAY_CUDA_ERROR_CHECK(hipDeviceSynchronize());
}

}  // namespace detray
