#include "hip/hip_runtime.h"
/** Detray library, part of the ACTS project (R&D line)
 *
 * (c) 2022 CERN for the benefit of the ACTS project
 *
 * Mozilla Public License Version 2.0
 */

#include "benchmark_propagator_cuda_kernel.hpp"
#include "detray/definitions/cuda_definitions.hpp"

namespace detray {

template <typename stepper_policy_t>
__global__ void propagator_benchmark_kernel(
    detector_view<detector_host_type> det_data,
    vecmem::data::vector_view<free_track_parameters> tracks_data,
    vecmem::data::jagged_vector_view<intersection_t> candidates_data) {

    int gid = threadIdx.x + blockIdx.x * blockDim.x;

    detector_device_type det(det_data);
    vecmem::device_vector<free_track_parameters> tracks(tracks_data);
    vecmem::jagged_device_vector<intersection_t> candidates(candidates_data);

    if (gid >= tracks.size()) {
        return;
    }

    // Set the magnetic field
    const vector3 B{0, 0, 2 * unit_constants::T};
    field_type B_field(B);

    // Create RK stepper
    rk_stepper<stepper_policy_t> s(B_field);

    // Create navigator
    navigator_device_type n(det);

    // Create propagator
    propagator_device_type<stepper_policy_t> p(std::move(s), std::move(n));

    // Create the propagator state
    typename propagator_device_type<stepper_policy_t>::state p_state(
        tracks.at(gid), actor_chain<>::state{}, candidates.at(gid));

    // Run propagation
    p.propagate(p_state);
}

template <typename stepper_policy_t>
void propagator_benchmark(
    detector_view<detector_host_type> det_data,
    vecmem::data::vector_view<free_track_parameters>& tracks_data,
    vecmem::data::jagged_vector_view<intersection_t>& candidates_data) {

    constexpr int thread_dim = 2 * WARP_SIZE;
    int block_dim = tracks_data.size() / thread_dim + 1;

    // run the test kernel
    propagator_benchmark_kernel<stepper_policy_t>
        <<<block_dim, thread_dim>>>(det_data, tracks_data, candidates_data);

    // cuda error check
    DETRAY_CUDA_ERROR_CHECK(hipGetLastError());
    DETRAY_CUDA_ERROR_CHECK(hipDeviceSynchronize());
}

template void propagator_benchmark<always_init>(
    detector_view<detector_host_type>,
    vecmem::data::vector_view<free_track_parameters>&,
    vecmem::data::jagged_vector_view<intersection_t>&);

template void propagator_benchmark<stepper_default_policy>(
    detector_view<detector_host_type>,
    vecmem::data::vector_view<free_track_parameters>&,
    vecmem::data::jagged_vector_view<intersection_t>&);

template void propagator_benchmark<stepper_rk_correction_policy>(
    detector_view<detector_host_type>,
    vecmem::data::vector_view<free_track_parameters>&,
    vecmem::data::jagged_vector_view<intersection_t>&);

}  // namespace detray