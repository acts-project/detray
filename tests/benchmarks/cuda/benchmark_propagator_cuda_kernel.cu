#include "hip/hip_runtime.h"
/** Detray library, part of the ACTS project (R&D line)
 *
 * (c) 2022 CERN for the benefit of the ACTS project
 *
 * Mozilla Public License Version 2.0
 */

#include "benchmark_propagator_cuda_kernel.hpp"
#include "detray/definitions/cuda_definitions.hpp"

namespace detray {

__global__ void __launch_bounds__(256, 4) propagator_benchmark_kernel(
    typename detector_host_type::view_type det_data,
    covfie::field_view<bfield::const_bknd_t> field_data,
    vecmem::data::vector_view<free_track_parameters<transform3>> tracks_data,
    vecmem::data::jagged_vector_view<intersection_t> candidates_data,
    const propagate_option opt) {

    int gid = threadIdx.x + blockIdx.x * blockDim.x;

    detector_device_type det(det_data);
    vecmem::device_vector<free_track_parameters<transform3>> tracks(
        tracks_data);
    vecmem::jagged_device_vector<intersection_t> candidates(candidates_data);

    if (gid >= tracks.size()) {
        return;
    }

    // Create RK stepper
    rk_stepper_type s;

    // Create navigator
    navigator_device_type n;

    // Create propagator
    propagator_device_type p(std::move(s), std::move(n));

    parameter_transporter<transform3>::state transporter_state{};
    pointwise_material_interactor<transform3>::state interactor_state{};
    parameter_resetter<transform3>::state resetter_state{};

    // Create the actor states
    auto actor_states =
        tie(transporter_state, interactor_state, resetter_state);
    // Create the propagator state
    propagator_device_type::state p_state(tracks.at(gid), field_data, det,
                                          candidates.at(gid));

    // Run propagation
    if (opt == propagate_option::e_unsync) {
        p.propagate(p_state, actor_states);
    } else if (opt == propagate_option::e_sync) {
        p.propagate_sync(p_state, actor_states);
    }
}

void propagator_benchmark(
    typename detector_host_type::view_type det_data,
    covfie::field_view<bfield::const_bknd_t> field_data,
    vecmem::data::vector_view<free_track_parameters<transform3>>& tracks_data,
    vecmem::data::jagged_vector_view<intersection_t>& candidates_data,
    const propagate_option opt) {

    constexpr int thread_dim = 256;
    int block_dim =
        static_cast<int>(tracks_data.size() + thread_dim - 1) / thread_dim;

    // run the test kernel
    propagator_benchmark_kernel<<<block_dim, thread_dim>>>(
        det_data, field_data, tracks_data, candidates_data, opt);

    // cuda error check
    DETRAY_CUDA_ERROR_CHECK(hipGetLastError());
    DETRAY_CUDA_ERROR_CHECK(hipDeviceSynchronize());
}

}  // namespace detray
