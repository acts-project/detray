#include "hip/hip_runtime.h"
/** Detray library, part of the ACTS project (R&D line)
 *
 * (c) 2025 CERN for the benefit of the ACTS project
 *
 * Mozilla Public License Version 2.0
 */

// Project include(s)
#include "detray/benchmarks/device/cuda/propagator.hpp"
#include "detray/definitions/detail/cuda_definitions.hpp"

// CUDA include(s)
#include <hip/hip_cooperative_groups.h>

#include <cuda/barrier>

namespace detray {

template <typename navigator_t, typename stepper_t, typename actor_chain_t>
struct propagation_queue {
    using barrier_type = ::cuda::barrier<::cuda::thread_scope_block>;
    using data_buffer_type =
        vecmem::device_vector<propagation_state<navigator_t, stepper_t>>;
    using slot_type = std::array<int, 2>;

    static constexpr int n_slots{2};
    static constexpr int queue_length{256};
    static constexpr int buffer_size{n_slots * queue_length};

    __device__ __shared__ barrier_type m_barriers[4];

    cooperative_groups::thread_group m_group;

    // Data to work on, shared between all blocks!
    data_buffer_type m_data;

    std::array<slot_type, n_slots> m_slots;

    DETRAY_DEVICE propagation_queue(const data_buffer_type data)
        : m_data{data} {

        // Block-wide init
        if (block.thread_rank() == 0) {
            printf("This is block %d\n", blockIdx.x);
            // Offset for this block's data into the state containers
            const int data_offset{buffer_size * static_cast<int>(blockIdx.x)};

            // Number of elements this block handles (comes in chunks of 512
            // tracks)
            const int n_tracks{math::min(
                static_cast<int>(m_data.size()) - data_offset, buffer_size)};

            m_slots[0] = {data_offset,
                          data_offset + math::min(n_tracks, queue_length)};
            m_slots[1] = {data_offset + queue_length,
                          data_offset + queue_length +
                              math::min(math::max(0, n_tracks - queue_length),
                                        queue_length)};

            printf("Block %d slot 0: [%d, %d]\n", blockIdx.x, m_slots[0][0],
                   m_slots[0][1]);
            printf("Block %d slot 1: [%d, %d]\n", blockIdx.x, m_slots[1][0],
                   m_slots[1][1]);
        }
        block.sync();

        if (block.thread_rank() < 4) {
            init(m_barriers + block.thread_rank(), block.size());
        }
        block.sync();
    }

    propagation_queue(const propagation_queue &data) = default;

    DETRAY_DEVICE constexpr void set_group(
        cooperative_groups::thread_group group) {
        m_group = group;
    }

    DETRAY_DEVICE constexpr cooperative_groups::thread_group get_group() const {
        return m_group;
    }

    DETRAY_DEVICE constexpr decltype(auto) navigation_state(
        const int step) const {
        return get_data(step)._navigation;
    }

    DETRAY_DEVICE constexpr decltype(auto) stepper_state(const int step) const {
        return get_data(step)._stepping;
    }

    DETRAY_DEVICE constexpr int slot_index(const int step) const {
        return step % 2;
    }

    DETRAY_DEVICE constexpr slot_type get_slot(const int step) const {
        return m_slots[slot_index(step)];
    }

    DETRAY_DEVICE constexpr int get_data_index(const int step) const {
        return m_group.thread_rank() + get_slot(step)[0];
    }

    DETRAY_DEVICE constexpr bool is_active(const int step) const {
        return get_data_index(step) < get_slot(step)[1];
    }

    DETRAY_DEVICE constexpr bool is_alive(const int step) const {
        return (is_active(step) && get_data(step)._navigation.is_alive());
    }

    DETRAY_DEVICE constexpr decltype(auto) get_data(const int step) const {
        return m_data[get_data_index(step)];
    }

    DETRAY_DEVICE constexpr decltype(auto) get_data(const int step) {
        return m_data[get_data_index(step)];
    }

    DETRAY_DEVICE constexpr decltype(auto) get_stepping_barrier(
        const int step) {
        return m_barriers[slot_index(step)];
    }

    DETRAY_DEVICE constexpr decltype(auto) get_navigation_barrier(
        const int step) {
        return m_barriers[slot_index(step) + 2];
    }

    DETRAY_DEVICE inline barrier_type::arrival_token wait_for_navigation(
        const int step) {
        printf(
            "Thread %d (Step): Wait for navigation (slot: %d, iteration %d)\n",
            m_group.thread_rank(), slot_index(step), step);
        // Wait until the navigation triggers stepping
        return get_stepping_barrier(step).arrive_and_wait();
    }

    DETRAY_DEVICE inline barrier_type::arrival_token wait_for_stepping(
        const int step) {
        printf("Thread %d (Nav): Wait for stepping (slot: %d, iteration %d)\n",
               m_group.thread_rank(), slot_index(step), step);
        // Wait until the stepper triggers navigation
        return get_navigation_barrier(step).arrive_and_wait();
    }

    DETRAY_DEVICE inline barrier_type::arrival_token trigger_navigation(
        const int step) {
        printf(
            "Thread %d (Step): Trigger navigation (slot: %d, iteration %d)\n",
            m_group.thread_rank(), slot_index(step), step);
        return get_navigation_barrier(step).arrive();
    }

    DETRAY_DEVICE inline barrier_type::arrival_token trigger_stepping(
        const int step) {
        printf("Thread %d (Nav): Trigger stepper (slot: %d, iteration %d)\n",
               m_group.thread_rank(), slot_index(step), step);
        return get_stepping_barrier(step).arrive();
    }

    DETRAY_DEVICE inline barrier_type::arrival_token finish_navigation(
        const int step) {
        printf(
            "Thread %d (Nav): Navigation finished (slot: %d, iteration %d)\n",
            m_group.thread_rank(), slot_index(step), step);
        return get_navigation_barrier(step).arrive_and_drop();
    }

    DETRAY_DEVICE inline barrier_type::arrival_token finish_stepping(
        const int step) {
        printf("Thread %d (Step): Stepping finished (slot: %d, iteration %d)\n",
               m_group.thread_rank(), slot_index(step), step);
        return get_stepping_barrier(step).arrive_and_drop();
    }
};

/// Initialize the stepper, navigator and actor states on device
template <typename navigator_t, typename stepper_t, typename actor_chain_t>
__global__ void __launch_bounds__(256, 4) propagation_init_kernel(
    const propagation::config *pinned_cfg,
    typename navigator_t::detector_type *pinned_detector_ptr,
    typename navigator_t::detector_type::view_type det_view,
    typename stepper_t::magnetic_field_type field_view,
    vecmem::data::vector_view<typename stepper_t::free_track_parameters_type>
        tracks_view,
    vecmem::data::vector_view<propagation_state<navigator_t, stepper_t>>
        propagation_states_view,
    vecmem::data::vector_view<typename actor_chain_t::state_tuple>
        actor_states_view,
    typename actor_chain_t::state_tuple *pinned_actor_state_ptr) {

    using detector_device_t = typename navigator_t::detector_type;
    using algebra_t = typename detector_device_t::algebra_type;

    using track_t = typename stepper_t::free_track_parameters_type;
    using stepper_state_t = typename stepper_t::state;
    using navigation_state_t = typename navigator_t::state;
    using propagation_state_t = propagation_state<navigator_t, stepper_t>;

    const vecmem::device_vector<track_t> tracks(tracks_view);
    vecmem::device_vector<propagation_state_t> propagation_states(
        propagation_states_view);
    vecmem::device_vector<typename actor_chain_t::state_tuple> actor_states(
        actor_states_view);

    constexpr navigator_t navigator{};
    constexpr actor_chain_t run_actors{};

    typename detector_device_t::geometry_context gctx{};

    // Setup some small amount of data in shared memory
    __shared__ navigation::config nav_cfg[1];
    __shared__ typename actor_chain_t::state_tuple actor_states_sh[1];

    auto block = cooperative_groups::this_thread_block();
    if (block.thread_rank() == 0) {
        nav_cfg[0] = pinned_cfg->navigation;
        actor_states_sh[0] = *pinned_actor_state_ptr;
    }
    block.sync();

    /// Every thread initializes one set of states
    int gid = threadIdx.x + blockIdx.x * blockDim.x;
    if (gid >= tracks.size()) {
        return;
    }

    // Create the stepper state:
    // The track gets copied into the stepper state, so that the
    // original track sample vector remains unchanged
    propagation_state_t &propagation_state = propagation_states.at(gid);
    stepper_state_t &stepping = propagation_state._stepping;
    stepping = stepper_state_t{tracks.at(gid), field_view};

    // Create navigator state
    navigation_state_t &navigation = propagation_state._navigation;
    navigation = navigation_state_t{pinned_detector_ptr};

    // Temporary propagation state
    propagation_state_t propagation{stepping, navigation, gctx};

    // Create the actor states on a fresh copy
    auto actor_states_ref =
        actor_chain_t::setup_actor_states(actor_states.at(gid));
    actor_states_ref = actor_states_sh[0];

    // Initialize the navigation
    navigator.init(stepping(), navigation, *nav_cfg, gctx);
    assert(navigation.is_alive());

    // Run all registered actors/aborters
    run_actors(actor_states_ref, propagation);
    assert(!stepping().is_invalid());
    assert(!stepping.bound_params().is_invalid());

    // Update the navigation information, in case the actors changed the track
    navigator.update(stepping(), navigation, *nav_cfg, gctx);
    assert(navigation.is_alive());
}

template <typename collection_t>
__device__ std::size_t scan(collection_t &c) {

    for (std::size_t i = 0; i < c.size(); ++i) {
    }
}

/// Run the stepper
template <typename navigator_t, typename stepper_t, typename actor_chain_t>
__device__ inline void take_step(
    const stepping::config *cfg, cooperative_groups::thread_group group,
    propagation_queue<navigator_t, stepper_t, actor_chain_t> queue
    // vecmem::device_vector<unsigned int> stepper_res_view,
    // vecmem::device_vector<unsigned int> navigator_res_view
) {

    using barrier_t = ::cuda::barrier<::cuda::thread_scope_block>;

    using detector_t = typename navigator_t::detector_type;
    using algebra_t = typename detector_t::algebra_type;
    using scalar_t = dscalar<algebra_t>;

    using stepper_state_t = typename stepper_t::state;
    using navigation_state_t = typename navigator_t::state;
    using propagation_state_t = propagation_state<navigator_t, stepper_t>;

    constexpr stepper_t stepper{};
    constexpr bool reset_stepsize{true};

    queue.set_group(group);
    vecmem::device_vector<propagation_state_t> propagation_states =
        queue.data();

    assert(cfg != nullptr);
    // assert(static_cast<int>(propagation_states.size()) > 512 * blockIdx.x);

    // Block-local thread index
    const int thread_idx =
        cooperative_groups::this_thread_block().thread_rank();

    if (queue.is_active(0)) {
        printf("Thread %d (Step): Has queue: 0\n", thread_idx);
        if (queue.is_alive(0)) {
            printf("Thread %d (Step): Navigation alive (queue: 0)\n",
                   thread_idx);
        }
    }
    if (queue.is_active(1)) {
        printf("Thread %d (Step): Has queue: 1\n", thread_idx);
        if (queue.is_alive(0)) {
            printf("Thread %d (Step): Navigation alive (queue: 1)\n",
                   thread_idx);
        }
    }
    if (!queue.is_active(0) && !queue.is_active(1)) {
        printf(
            "Thread %d (Step): Has NO queue: (state idx %d [%d, "
            "%d])\n",
            thread_idx, state_idx_0, queues[0][0], queues[0][1]);
        printf(
            "Thread %d (Step): Has NO queue: (state idx %d [%d, "
            "%d])\n",
            thread_idx, state_idx_1, queues[1][0], queues[1][1]);
    }

    // Run while either at least one of the two navigation streams monitored by
    // this thread is alive
    int i{0};
    while (i < 1000) {
        queue.wait_for_navigation(i);

        printf("Thread %d (Step): Navigation ready (iteration %d)\n",
               thread_idx, i);
        if (queue.is_alive(i)) {
            navigation_state_t &navigation = queue.navigation_state(i);

            if (navigation.is_alive()) {
                printf("Thread %d (Step): Take step ready (iteration %d)\n",
                       thread_idx, i);

                stepper_state_t &stepping = queue.stepper_state(i);
                const auto &track = stepping();

                const auto vol = navigation.get_volume();
                const material<scalar_t> *vol_mat_ptr =
                    vol.has_material() ? vol.material_parameters(track.pos())
                                       : nullptr;

                stepper.step(navigation(), stepping, *cfg, reset_stepsize,
                             vol_mat_ptr);

                typename stepper_t::policy_type{}(stepping.policy_state(),
                                                  queue.get_data(i));
            }
        }

        if (!queue.is_alive(i)) {
            queue.finish_stepping(i);
        } else {
            barrier_t::arrival_token token = queue.trigger_navigation(i);
        }
        ++i;
    }
    if (i >= 1000) {
        printf(
            "Thread %d, block %d  (Step): Drop - Reached hard limit, "
            "(iteration %d)\n",
            thread_idx, blockIdx.x, i);
        needs_navigation[0].arrive_and_drop();
        needs_navigation[1].arrive_and_drop();
    }
    // needs_stepping[0].arrive_and_drop();
    // needs_stepping[1].arrive_and_drop();

    printf("Thread %d, block %d (Step): Stepping finished\n", thread_idx,
           blockIdx.x);
}

/// Run the navigator
template <typename navigator_t, typename stepper_t, typename actor_chain_t>
__device__ void navigate(
    const navigation::config *cfg, cooperative_groups::thread_group group,
    propagation_queue<navigator_t, stepper_t, actor_chain_t> queue
    // vecmem::device_vector<unsigned int> stepper_res_view,
    // vecmem::device_vector<unsigned int> navigator_res_view
) {

    using barrier_t = ::cuda::barrier<::cuda::thread_scope_block>;

    using detector_t = typename navigator_t::detector_type;
    using stepper_state_t = typename stepper_t::state;
    using navigation_state_t = typename navigator_t::state;

    queue.set_group(group);
    vecmem::device_vector<propagation_state_t> propagation_states =
        queue.data();

    assert(cfg != nullptr);
    assert(static_cast<int>(propagation_states.size()) > 512 * blockIdx.x);

    typename detector_t::geometry_context gctx{};
    constexpr navigator_t navigator{};

    // Block-local thread index (adjusted for the start of the warp
    // specialization)
    const int thread_idx =
        cooperative_groups::this_thread_block().thread_rank() - blockDim.x / 2;
    assert(thread_idx >= 0);

    // Wait for all threads in the block to arrive for stepping
    printf("Thread %d (Nav): Needs stepping (queue: 0)\n", thread_idx);
    barrier_t::arrival_token token1 = needs_stepping[0].arrive();
    printf("Thread %d (Nav): Needs stepping (queue: 1)\n", thread_idx);
    barrier_t::arrival_token token2 = needs_stepping[1].arrive();

    // Does this thread have work in either queue ?
    const int state_idx_0{thread_idx + queues[0][0]};
    const int state_idx_1{thread_idx + queues[1][0]};

    const std::array<bool, 2> has_queue{state_idx_0 < queues[0][1],
                                        state_idx_1 < queues[1][1]};

    /*if (blockIdx.x == 1) {
        printf(
            "Thread %d, block %d (Nav): queue 0 %d (state idx %d [%d, "
            "%d])\n",
            thread_idx, blockIdx.x, has_queue[0], state_idx_0, queues[0][0],
            queues[0][1]);
        printf(
            "Thread %d, block %d (Nav): queue 1 %d (state idx %d [%d, "
            "%d])\n",
            thread_idx, blockIdx.x, has_queue[1], state_idx_1, queues[1][0],
            queues[1][1]);
    }*/

    if (has_queue[0]) {
        printf("Thread %d (Nav): Has queue: 0\n", thread_idx);
        if (propagation_states.at(state_idx_0)._navigation.is_alive()) {
            printf("Thread %d (Nav): Navigation alive (queue: 0)\n",
                   thread_idx);
        }
    }
    if (has_queue[1]) {
        printf("Thread %d (Nav): Has queue: 1\n", thread_idx);
        if (propagation_states.at(state_idx_1)._navigation.is_alive()) {
            printf("Thread %d (Nav): Navigation alive (queue: 1)\n",
                   thread_idx);
        }
    }
    if (!has_queue[0] && !has_queue[1]) {
        printf(
            "Thread %d (Nav): Has NO queue: (state idx %d [%d, "
            "%d])\n",
            thread_idx, state_idx_0, queues[0][0], queues[0][1]);
        printf(
            "Thread %d (Nav): Has NO queue: (state idx %d [%d, "
            "%d])\n",
            thread_idx, state_idx_1, queues[1][0], queues[1][1]);
    }

    // Run while either at least one of the two navigation streams monitored by
    // this thread is alive
    int i{0};
    while (i < 1000) {
        // Queue for this iteration
        const int queue_idx{i % 2};
        const std::array<int, 2> &current_queue{queues[queue_idx]};
        // Which state to get
        auto state_idx = thread_idx + current_queue[0];

        printf("Thread %d (Nav): Wait for stepping (queue: %d, iteration %d)\n",
               thread_idx, queue_idx, i);
        // Wait for stepping to fill the queue
        needs_navigation[queue_idx].arrive_and_wait();
        printf(
            "Thread %d (Nav): Run state index %d (queue: %d, iteration %d)\n",
            thread_idx, state_idx, queue_idx, i);

        // If eligible to do work: navigate
        if (state_idx < current_queue[1]) {

            navigation_state_t &navigation =
                propagation_states.at(state_idx)._navigation;

            if (navigation.is_alive()) {
                const auto &stepper_state =
                    propagation_states.at(state_idx)._stepping;
                navigator.update(stepper_state(), navigation, *cfg, gctx);
            }
        }

        if (!(has_queue[queue_idx] &&
              propagation_states.at(state_idx)._navigation.is_alive())) {
            printf(
                "Thread %d (Nav): Drop out of propagation, (queue: %d, "
                "iteration %d)\n",
                thread_idx, queue_idx, i);
            // No more stepping needed
            needs_stepping[queue_idx].arrive_and_drop();

            // Unblock stepping to exit
            // needs_navigation[queue_idx].arrive_and_drop();
        } else {
            printf(
                "Thread %d (Nav): Needs stepping (queue: %d, iteration %d)\n",
                thread_idx, queue_idx, i);
            barrier_t::arrival_token token = needs_stepping[queue_idx].arrive();
        }
        ++i;
    }
    if (i >= 1000) {
        printf(
            "Thread %d, block %d  (Nav): Drop - Reached hard limit, (iteration "
            "%d)\n",
            thread_idx, blockIdx.x, i);
        needs_stepping[0].arrive_and_drop();
        needs_stepping[1].arrive_and_drop();
    }

    // needs_navigation[0].arrive_and_drop();
    // needs_navigation[1].arrive_and_drop();

    printf("Thread %d, block %d (Nav): Navigation finished\n", thread_idx,
           blockIdx.x);
}

/// Specialize the warps to run stepping, navigation and actors independently
template <typename navigator_t, typename stepper_t, typename actor_chain_t>
__global__ void __launch_bounds__(256, 4) propagation_kernel(
    const propagation::config *pinned_cfg,
    const typename navigator_t::detector_type *pinned_detector_ptr,
    vecmem::data::vector_view<propagation_state<navigator_t, stepper_t>>
        propagation_states_view,
    // vecmem::data::vector_view<unsigned int> stepper_res_view,
    // vecmem::data::vector_view<unsigned int> navigator_res_view,
    vecmem::data::vector_view<typename actor_chain_t::state_tuple>
        actor_states_view) {

    using barrier_t = ::cuda::barrier<::cuda::thread_scope_block>;
    using algebra_t = typename navigator_t::detector_type::algebra_type;

    /// Register the actor types
    const actor_chain_t run_actors{};

    vecmem::device_vector<propagation_state<navigator_t, stepper_t>>
        propagation_states(propagation_states_view);
    // vecmem::device_vector<unsigned int> stepper_results(stepper_res_view);

    int gid = threadIdx.x + blockIdx.x * blockDim.x;

    assert(blockDim.y == blockDim.z == 1);

    __shared__ stepping::config step_cfg[1];
    __shared__ navigation::config nav_cfg[1];

    propagation_queue<navigator_t, stepper_t, actor_chain_t> queue{
        propagation_states};

    // In each iteration, handles only half of the tracks (two queues: try to
    // avoid stalls)
    // constexpr int queue_length{256};
    /*constexpr int queue_length{32};

    // Map the two queues onto the state containers
    using queue_t = std::array<int, 2>;

    __shared__ queue_t queues[2];

    auto block = cooperative_groups::this_thread_block();
    if (block.thread_rank() == 0) {

        printf("This is block %d\n", blockIdx.x);
        step_cfg[0] = pinned_cfg->stepping;
        nav_cfg[0] = pinned_cfg->navigation;

        // Offset for this block's data into the state containers
        const int trk_offset{512 * static_cast<int>(blockIdx.x)};
        // Number of tracks this block handles (comes in chunks of 512 tracks)
        const int n_tracks{math::min(
            static_cast<int>(propagation_states.size()) - trk_offset, 512)};

        const queue_t first_queue = {
            trk_offset, trk_offset + math::min(n_tracks, queue_length)};
        const queue_t second_queue = {
            trk_offset + queue_length,
            trk_offset + queue_length +
                math::min(math::max(0, n_tracks - queue_length), queue_length)};

        printf("Block %d queue 0: [%d, %d]\n", blockIdx.x, first_queue[0],
               first_queue[1]);
        printf("Block %d queue 1: [%d, %d]\n", blockIdx.x, second_queue[0],
               second_queue[1]);

        queues[0] = first_queue;
        queues[1] = second_queue;
    }
    block.sync();*/

    // vecmem::device_vector<unsigned int>
    // navigation_results(navigator_res_view);

    /*if (gid < navigator_states_view.size()) {
        const auto dir = propagation_states.at(gid)().dir();
        printf("%f: [%f, %f, %f]\n", propagation_states._navigation.at(gid)(),
    dir[0], dir[1], dir[2]);
    }

    // bar[0] and bar[1] track if buffers buffer_0 and buffer_1 are ready to be
    // filled, while bar[2] and bar[3] track if buffers buffer_0 and buffer_1
    // are filled-in respectively
    __shared__ barrier_t bar[4];

    if (block.thread_rank() < 4) {
        init(bar + block.thread_rank(), block.size());
    }
    block.sync();*/

    const int split_idx = blockDim.x / 2;
    if (block.thread_rank() < split_idx) {
        printf("Thread %d (%d) in block %d doing stepping\n", gid,
               block.thread_rank(), blockIdx.x);
        take_step<navigator_t, stepper_t, actor_chain_t>(step_cfg, queue);
    } else {
        printf("Thread %d (%d) in block %d doing navigation\n", gid,
               block.thread_rank(), blockIdx.x);
        navigate<navigator_t, stepper_t, actor_chain_t>(nav_cfg, queue);
    }
}

typename propagation::config *setup_config(
    const typename propagation::config *input_config) {

    // Copy the config to the device
    propagation::config *pinned_config_ptr{nullptr};

    DETRAY_CUDA_ERROR_CHECK(hipHostAlloc((void **)&pinned_config_ptr,
                                          sizeof(propagation::config),
                                          hipHostMallocPortable));

    DETRAY_CUDA_ERROR_CHECK(hipMemcpy(pinned_config_ptr, input_config,
                                       sizeof(propagation::config),
                                       hipMemcpyHostToDevice));

    return pinned_config_ptr;
}

void release_config(typename propagation::config *pinned_config_ptr) {
    DETRAY_CUDA_ERROR_CHECK(hipHostFree(pinned_config_ptr));
}

template <typename device_detector_t>
device_detector_t *setup_device_detector(
    typename device_detector_t::view_type det_view) {

    // Build a device detector type (the interal pointers and capacities refer
    // to the already allocated vecmem device buffers)
    device_detector_t device_det{det_view};

    // Allocate global memory space for the device detector to be shared by
    // kernels
    device_detector_t *pinned_detector_ptr{nullptr};

    DETRAY_CUDA_ERROR_CHECK(hipHostAlloc((void **)&pinned_detector_ptr,
                                          sizeof(device_detector_t),
                                          hipHostMallocPortable));

    DETRAY_CUDA_ERROR_CHECK(hipMemcpy(pinned_detector_ptr, &device_det,
                                       sizeof(device_detector_t),
                                       hipMemcpyHostToDevice));

    return pinned_detector_ptr;
}

template <typename device_detector_t>
void release_device_detector(device_detector_t *pinned_detector_ptr) {
    DETRAY_CUDA_ERROR_CHECK(hipHostFree(pinned_detector_ptr));
}

template <typename actor_chain_t>
typename actor_chain_t::state_tuple *setup_actor_states(
    typename actor_chain_t::state_tuple *input_actor_states) {

    // Copy the actor state blueprint to the device
    using actor_state_t = typename actor_chain_t::state_tuple;
    actor_state_t *pinned_actor_state_ptr{nullptr};

    DETRAY_CUDA_ERROR_CHECK(hipHostAlloc((void **)&pinned_actor_state_ptr,
                                          sizeof(actor_state_t),
                                          hipHostMallocPortable));

    DETRAY_CUDA_ERROR_CHECK(
        hipMemcpy(pinned_actor_state_ptr, input_actor_states,
                   sizeof(actor_state_t), hipMemcpyHostToDevice));

    return pinned_actor_state_ptr;
}

template <typename actor_chain_t>
void release_actor_states(
    typename actor_chain_t::state_tuple *pinned_actor_state_ptr) {
    DETRAY_CUDA_ERROR_CHECK(hipHostFree(pinned_actor_state_ptr));
}

template <typename navigator_t, typename stepper_t, typename actor_chain_t>
void run_propagation_init_kernel(
    const propagation::config *pinned_cfg,
    typename navigator_t::detector_type *pinned_detector_ptr,
    typename navigator_t::detector_type::view_type det_view,
    typename stepper_t::magnetic_field_type field_view,
    vecmem::data::vector_view<typename stepper_t::free_track_parameters_type>
        tracks_view,
    vecmem::data::vector_view<propagation_state<navigator_t, stepper_t>>
        propagation_states_view,
    vecmem::data::vector_view<typename actor_chain_t::state_tuple>
        actor_states_view,
    typename actor_chain_t::state_tuple *pinned_actor_state_ptr) {

    constexpr int thread_dim = 256;
    int block_dim = (tracks_view.size() + thread_dim - 1) / thread_dim;

    // run the test kernel
    propagation_init_kernel<navigator_t, stepper_t, actor_chain_t>
        <<<block_dim, thread_dim>>>(
            pinned_cfg, pinned_detector_ptr, det_view, field_view, tracks_view,
            propagation_states_view, actor_states_view, pinned_actor_state_ptr);

    // cuda error check
    DETRAY_CUDA_ERROR_CHECK(hipGetLastError());
    DETRAY_CUDA_ERROR_CHECK(hipDeviceSynchronize());
}

template <typename navigator_t, typename stepper_t, typename actor_chain_t>
void run_propagation_kernel(
    const propagation::config *pinned_cfg,
    const typename navigator_t::detector_type *pinned_detector_ptr,
    vecmem::data::vector_view<propagation_state<navigator_t, stepper_t>>
        propagation_states_view,
    vecmem::data::vector_view<unsigned int> stepper_res_view,
    vecmem::data::vector_view<unsigned int> navigator_res_view,
    vecmem::data::vector_view<typename actor_chain_t::state_tuple>
        actor_states_view) {

    constexpr int thread_dim = 32;
    // One block handles 512 tracks
    int block_dim = (propagation_states_view.size() + 2 * thread_dim - 1) /
                    (2 * thread_dim);

    std::cout << "# Tracks: " << propagation_states_view.size() << std::endl;
    std::cout << "# threads per block: " << thread_dim
              << "\n# blocks: " << block_dim
              << "\n# threads: " << thread_dim * block_dim << std::endl;

    // run the propagation loop
    propagation_kernel<navigator_t, stepper_t, actor_chain_t>
        <<<block_dim, thread_dim>>>(pinned_cfg, pinned_detector_ptr,
                                    propagation_states_view, actor_states_view);

    // cuda error check
    DETRAY_CUDA_ERROR_CHECK(hipGetLastError());
    DETRAY_CUDA_ERROR_CHECK(hipDeviceSynchronize());
}

/// Macro declaring the template instantiations for the different detector types
#define DECLARE_PROPAGATOR(METADATA, CHAIN, FIELD)                          \
                                                                            \
    template void run_propagation_init_kernel<                              \
        navigator_type<METADATA>, stepper_type<METADATA, FIELD>,            \
        CHAIN<detector<METADATA>::algebra_type>>(                           \
        const propagation::config *,                                        \
        detector<METADATA, device_container_types> *,                       \
        typename detector<METADATA>::view_type, covfie::field_view<FIELD>,  \
        vecmem::data::vector_view<                                          \
            free_track_parameters<detector<METADATA>::algebra_type>>,       \
        vecmem::data::vector_view<propagation_state<                        \
            navigator_type<METADATA>, stepper_type<METADATA, FIELD>>>,      \
        vecmem::data::vector_view<                                          \
            typename CHAIN<detector<METADATA>::algebra_type>::state_tuple>, \
        typename CHAIN<detector<METADATA>::algebra_type>::state_tuple *);   \
                                                                            \
    template void run_propagation_kernel<                                   \
        navigator_type<METADATA>, stepper_type<METADATA, FIELD>,            \
        CHAIN<detector<METADATA>::algebra_type>>(                           \
        const propagation::config *,                                        \
        const detector<METADATA, device_container_types> *,                 \
        vecmem::data::vector_view<propagation_state<                        \
            navigator_type<METADATA>, stepper_type<METADATA, FIELD>>>,      \
        vecmem::data::vector_view<unsigned int>,                            \
        vecmem::data::vector_view<unsigned int>,                            \
        vecmem::data::vector_view<                                          \
            typename CHAIN<detector<METADATA>::algebra_type>::state_tuple>);

#define DECLARE_ACTOR_CHAIN_SETUP(METADATA, CHAIN)                           \
                                                                             \
    template typename CHAIN<detector<METADATA>::algebra_type>::state_tuple * \
    setup_actor_states<CHAIN<detector<METADATA>::algebra_type>>(             \
        typename CHAIN<detector<METADATA>::algebra_type>::state_tuple *);    \
                                                                             \
    template void                                                            \
    release_actor_states<CHAIN<detector<METADATA>::algebra_type>>(           \
        typename CHAIN<detector<METADATA>::algebra_type>::state_tuple *);

#define DECLARE_DETECTOR_ALLOCATION(METADATA)                                \
                                                                             \
    template detector<METADATA, device_container_types>                      \
        *setup_device_detector<detector<METADATA, device_container_types>>(  \
            typename detector<METADATA, device_container_types>::view_type); \
                                                                             \
    template void                                                            \
    release_device_detector<detector<METADATA, device_container_types>>(     \
        detector<METADATA, device_container_types> *);

DECLARE_PROPAGATOR(benchmarks::default_metadata, empty_chain, const_field_t)
DECLARE_PROPAGATOR(benchmarks::default_metadata, default_chain, const_field_t)

DECLARE_PROPAGATOR(benchmarks::toy_metadata, empty_chain, const_field_t)
DECLARE_PROPAGATOR(benchmarks::toy_metadata, default_chain, const_field_t)

// Declare only once per algebra type
DECLARE_ACTOR_CHAIN_SETUP(benchmarks::toy_metadata, empty_chain)
DECLARE_ACTOR_CHAIN_SETUP(benchmarks::toy_metadata, default_chain)

DECLARE_DETECTOR_ALLOCATION(benchmarks::default_metadata)
DECLARE_DETECTOR_ALLOCATION(benchmarks::toy_metadata)

}  // namespace detray
