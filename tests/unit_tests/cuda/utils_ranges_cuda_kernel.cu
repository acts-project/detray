#include "hip/hip_runtime.h"
/** Detray library, part of the ACTS project (R&D line)
 *
 * (c) 2022 CERN for the benefit of the ACTS project
 *
 * Mozilla Public License Version 2.0
 */

#include "detray/definitions/cuda_definitions.hpp"
#include "utils_ranges_cuda_kernel.hpp"

namespace detray {

__global__ void single_kernel(const dindex value, dindex* result) {

    // single view should ony add the value 'i' once
    for (auto i : detray::views::single(value)) {
        *result += i;
    }
}

void single(const dindex value, dindex& check) {
    dindex* result{nullptr};
    hipMallocManaged(&result, sizeof(dindex));
    *result = 0;

    // run the kernel
    single_kernel<<<1, 1>>>(value, result);

    // cuda error check
    DETRAY_CUDA_ERROR_CHECK(hipGetLastError());
    DETRAY_CUDA_ERROR_CHECK(hipDeviceSynchronize());

    check = *result;
    hipFree(result);
}

/*__global__ void sequence_single_kernel(
    vecmem::data::vector_view<dindex> check_data,
    vecmem::data::vector_view<dindex> single_data) {

    vecmem::device_vector<dindex> check(check_data);
    vecmem::device_vector<dindex> single(single_data);

    for (auto i : detray::views::iota(single[0])) {
        check[0] += i;
    }
}

void sequence_single(vecmem::data::vector_view<dindex>& check_data,
                     vecmem::data::vector_view<dindex>& single_data) {

    // run the kernel
    sequence_single_kernel<<<1, 1>>>(check_data, single_data);

    // cuda error check
    DETRAY_CUDA_ERROR_CHECK(hipGetLastError());
    DETRAY_CUDA_ERROR_CHECK(hipDeviceSynchronize());
}

__global__ void sequence_range_kernel(
    const darray<dindex, 2> range,
    vecmem::data::vector_view<dindex> check_data) {

    vecmem::device_vector<dindex> check(check_data);

    for (auto i : detray::views::iota(range)) {
        check.push_back(i);
    }
}

void sequence_range(const darray<dindex, 2> range,
                    vecmem::data::vector_view<dindex>& check_data) {

    // run the kernel
    sequence_range_kernel<<<1, 1>>>(range, check_data);

    // cuda error check
    DETRAY_CUDA_ERROR_CHECK(hipGetLastError());
    DETRAY_CUDA_ERROR_CHECK(hipDeviceSynchronize());
}

__global__ void enumerate_sequence_kernel(
    vecmem::data::vector_view<dindex> idx_data,
    vecmem::data::vector_view<unsigned int> uint_data,
    vecmem::data::vector_view<uint_holder> seq_data) {

    vecmem::device_vector<dindex> idx_vec(idx_data);
    vecmem::device_vector<unsigned int> uint_vec(uint_data);
    vecmem::device_vector<uint_holder> seq(seq_data);

    for (auto [i, v] : detray::views::enumerate(seq)) {
        idx_vec.push_back(i);
        uint_vec.push_back(v.ui);
    }
}

void enumerate_sequence(vecmem::data::vector_view<dindex>& idx_data,
                        vecmem::data::vector_view<unsigned int>& uint_data,
                        vecmem::data::vector_view<uint_holder>& seq_data) {

    // run the kernel
    enumerate_sequence_kernel<<<1, 1>>>(idx_data, uint_data, seq_data);

    // cuda error check
    DETRAY_CUDA_ERROR_CHECK(hipGetLastError());
    DETRAY_CUDA_ERROR_CHECK(hipDeviceSynchronize());
}

__global__ void iterate_range_kernel(vecmem::data::vector_view<int> check_data,
                                     vecmem::data::vector_view<int> seq_data,
                                     const size_t begin, const size_t end) {

    vecmem::device_vector<int> check(check_data);
    vecmem::device_vector<int> seq(seq_data);

    for (const auto& v :
         detray::ranges::subrange(seq, std::array<size_t, 2>{begin, end})) {
        check.push_back(v);
    }
}

void iterate_range(vecmem::data::vector_view<int>& check_data,
                   vecmem::data::vector_view<int>& seq_data,
                   const size_t& begin, const size_t& end) {

    // run the kernel
    iterate_range_kernel<<<1, 1>>>(check_data, seq_data, begin, end);

    // cuda error check
    DETRAY_CUDA_ERROR_CHECK(hipGetLastError());
    DETRAY_CUDA_ERROR_CHECK(hipDeviceSynchronize());
}*/

}  // namespace detray
