#include "hip/hip_runtime.h"
/** Detray library, part of the ACTS project (R&D line)
 *
 * (c) 2022 CERN for the benefit of the ACTS project
 *
 * Mozilla Public License Version 2.0
 */

#include <vecmem/containers/device_vector.hpp>

#include "detray/definitions/cuda_definitions.hpp"
#include "rk_stepper_cuda_kernel.hpp"

namespace detray {

__global__ void bound_state_test_kernel(
    vecmem::data::vector_view<bound_track_parameters> out_param,
    const bound_track_parameters in_param, const vector3 B,
    const transform3 trf) {

    vecmem::device_vector<bound_track_parameters> out_param_cuda(out_param);

    mag_field_t mag_field(B);
    prop_state<crk_stepper_t::state, nav_state> propagation{
        crk_stepper_t::state(in_param, trf), nav_state{}};
    crk_stepper_t::state &crk_state = propagation._stepping;
    nav_state &n_state = propagation._navigation;

    // Decrease tolerance down to 1e-8
    crk_state.set_tolerance(rk_tolerance);

    // RK stepper and its state
    crk_stepper_t crk_stepper(mag_field);

    // Path length per turn
    scalar S = 2. * std::fabs(1. / in_param.qop()) / getter::norm(B) * M_PI;

    // Run stepper for one turn
    unsigned int max_steps = 1e4;
    for (unsigned int i = 0; i < max_steps; i++) {

        crk_state.set_constraint(S - crk_state.path_length());

        n_state._step_size = S;

        crk_stepper.step(propagation);

        if (std::abs(S - crk_state.path_length()) < 1e-6) {
            break;
        }
    }

    // Bound state after one turn propagation
    out_param_cuda[0] = crk_stepper.bound_state(propagation, trf);
}

void bound_state_test(
    vecmem::data::vector_view<bound_track_parameters> out_param,
    const bound_track_parameters in_param, const vector3 B,
    const transform3 trf) {

    constexpr int thread_dim = 1;
    constexpr int block_dim = 1;

    // run the test kernel
    bound_state_test_kernel<<<block_dim, thread_dim>>>(out_param, in_param, B,
                                                       trf);

    // cuda error check
    DETRAY_CUDA_ERROR_CHECK(hipGetLastError());
    DETRAY_CUDA_ERROR_CHECK(hipDeviceSynchronize());
}

}  // namespace detray