#include "hip/hip_runtime.h"
/** Detray library, part of the ACTS project (R&D line)
 *
 * (c) 2022 CERN for the benefit of the ACTS project
 *
 * Mozilla Public License Version 2.0
 */

#include <vecmem/containers/device_vector.hpp>

#include "detray/definitions/cuda_definitions.hpp"
#include "rk_stepper_cuda_kernel.hpp"

namespace detray {

__global__ void rk_stepper_test_kernel(
    vecmem::data::vector_view<free_track_parameters> tracks_data,
    const vector3 B) {

    int gid = threadIdx.x + blockIdx.x * blockDim.x;
    vecmem::device_vector<free_track_parameters> tracks(tracks_data);

    // Prevent overflow
    if (gid >= tracks.size()) {
        return;
    }

    // Define RK [constrained] stepper
    rk_stepper_t rk_stepper(B);
    crk_stepper_t crk_stepper(B);

    // Get a track
    auto &traj = tracks.at(gid);
    free_track_parameters c_traj(traj);

    // RK Stepping into forward direction
    prop_state<rk_stepper_t::state> propagation{rk_stepper_t::state{traj},
                                                nav_state{}};
    prop_state<crk_stepper_t::state> c_propagation{crk_stepper_t::state{c_traj},
                                                   nav_state{}};

    crk_stepper_t::state &crk_state = c_propagation._stepping;

    nav_state &n_state = propagation._navigation;
    nav_state &cn_state = c_propagation._navigation;

    // Forward direction
    crk_state.template set_constraint<step::constraint::e_user>(
        0.5 * unit_constants::mm);
    n_state._step_size = 1. * unit_constants::mm;
    cn_state._step_size = 1. * unit_constants::mm;
    for (unsigned int i_s = 0; i_s < rk_steps; i_s++) {
        rk_stepper.step(propagation);
        crk_stepper.step(c_propagation);
        crk_stepper.step(c_propagation);
    }

    // Backward direction
    // Roll the same track back to the origin
    n_state._step_size *= -1. * unit_constants::mm;
    cn_state._step_size = 1. * unit_constants::mm;
    for (unsigned int i_s = 0; i_s < rk_steps; i_s++) {
        rk_stepper.step(propagation);
        crk_stepper.step(c_propagation);
        crk_stepper.step(c_propagation);
    }
}

void rk_stepper_test(
    vecmem::data::vector_view<free_track_parameters> &tracks_data,
    const vector3 B) {

    constexpr int thread_dim = 2 * WARP_SIZE;
    constexpr int block_dim = theta_steps * phi_steps / thread_dim + 1;

    // run the test kernel
    rk_stepper_test_kernel<<<block_dim, thread_dim>>>(tracks_data, B);

    // cuda error check
    DETRAY_CUDA_ERROR_CHECK(hipGetLastError());
    DETRAY_CUDA_ERROR_CHECK(hipDeviceSynchronize());
}

}  // namespace detray