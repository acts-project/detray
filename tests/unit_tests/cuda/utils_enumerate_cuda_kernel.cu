#include "hip/hip_runtime.h"
/** Detray library, part of the ACTS project (R&D line)
 *
 * (c) 2022 CERN for the benefit of the ACTS project
 *
 * Mozilla Public License Version 2.0
 */

#include "detray/definitions/cuda_defs.hpp"
#include "utils_enumerate_cuda_kernel.hpp"

namespace detray {

__global__ void sequence_single_kernel(
    vecmem::data::vector_view<dindex> check_data,
    vecmem::data::vector_view<dindex> single_data) {

    vecmem::device_vector<dindex> check(check_data);
    vecmem::device_vector<dindex> single(single_data);

    for (auto i : sequence(single[0])) {
        check[0] += i;
    }
}

void sequence_single(vecmem::data::vector_view<dindex>& check_data,
                     vecmem::data::vector_view<dindex>& single_data) {

    // run the kernel
    sequence_single_kernel<<<1, 1>>>(check_data, single_data);

    // cuda error check
    DETRAY_CUDA_ERROR_CHECK(hipGetLastError());
    DETRAY_CUDA_ERROR_CHECK(hipDeviceSynchronize());
}

__global__ void sequence_range_kernel(
    const darray<dindex, 2> range,
    vecmem::data::vector_view<dindex> check_data) {

    vecmem::device_vector<dindex> check(check_data);

    for (auto i : sequence(range)) {
        check.push_back(i);
    }
}

void sequence_range(const darray<dindex, 2> range,
                    vecmem::data::vector_view<dindex>& check_data) {

    // run the kernel
    sequence_range_kernel<<<1, 1>>>(range, check_data);

    // cuda error check
    DETRAY_CUDA_ERROR_CHECK(hipGetLastError());
    DETRAY_CUDA_ERROR_CHECK(hipDeviceSynchronize());
}

__global__ void enumerate_sequence_kernel(
    vecmem::data::vector_view<dindex> idx_data,
    vecmem::data::vector_view<unsigned int> uint_data,
    vecmem::data::vector_view<uint_holder> seq_data) {

    vecmem::device_vector<dindex> idx_vec(idx_data);
    vecmem::device_vector<unsigned int> uint_vec(uint_data);
    vecmem::device_vector<uint_holder> seq(seq_data);

    for (auto [i, v] : enumerate(seq)) {
        idx_vec.push_back(i);
        uint_vec.push_back(v.ui);
    }
}

void enumerate_sequence(vecmem::data::vector_view<dindex>& idx_data,
                        vecmem::data::vector_view<unsigned int>& uint_data,
                        vecmem::data::vector_view<uint_holder>& seq_data) {

    // run the kernel
    enumerate_sequence_kernel<<<1, 1>>>(idx_data, uint_data, seq_data);

    // cuda error check
    DETRAY_CUDA_ERROR_CHECK(hipGetLastError());
    DETRAY_CUDA_ERROR_CHECK(hipDeviceSynchronize());
}

}  // namespace detray
