#include "hip/hip_runtime.h"
/** Detray library, part of the ACTS project (R&D line)
 *
 * (c) 2021 CERN for the benefit of the ACTS project
 *
 * Mozilla Public License Version 2.0
 */

#include <vecmem/containers/device_vector.hpp>

#include "definitions/cuda_defs.hpp"
#include "grids_grid2_cuda_kernel.cuh"

namespace detray {

/*---------------------------------------------------
  test function for grid data with replace populator
  ---------------------------------------------------*/

// test1 kernel declaration
template <typename grid_data_t>
__global__ void grid_test1_kernel(grid_data_t grid_data);

// test1 instantiation for replace populator
template void grid_test1<grid2r_replace_data>(grid2r_replace_data& grid_data);

// test2 function implementation
template <typename grid2_data_t>
void grid_test1(grid2_data_t& grid_data) {

    // auto& data_view = grid_data._data_serialized;
    const auto& axis0 = grid_data._axis_p0;
    const auto& axis1 = grid_data._axis_p1;

    int num_blocks = 1;
    int num_threads = axis0.bins() * axis1.bins();

    // run the kernel
    grid_test1_kernel<<<num_blocks, num_threads>>>(grid_data);

    // cuda error check
    DETRAY_CUDA_ERROR_CHECK(hipGetLastError());
    DETRAY_CUDA_ERROR_CHECK(hipDeviceSynchronize());
}

// test2 kernel implementation
template <typename grid_data_t>
__global__ void grid_test1_kernel(grid_data_t grid_data) {
    /*
    typename grid_data_t::populator_t::device_vector_t data_device(
        grid_data._data_serialized);
    const auto& axis0 = grid_data._axis_p0;
    const auto& axis1 = grid_data._axis_p1;

    auto& pt = data_device[threadIdx.x];

    auto x_interval = (axis0.max - axis0.min) / axis0.n_bins;
    auto y_interval = (axis1.max - axis1.min) / axis1.n_bins;

    pt = test::point3{axis0.min + threadIdx.x * x_interval,
                      axis1.min + threadIdx.x * y_interval, 0.5};
    */
}

/*---------------------------------------------------------------
  test function for grid data with complete and attach populator
  ---------------------------------------------------------------*/

// test2 kernel declaration
template <typename grid_data_t>
__global__ void grid_test2_kernel(grid_data_t grid_data);

// test2 instantiation for complete populator
template void grid_test2<grid2r_complete_data>(grid2r_complete_data& grid_data);

// test2 instantiation for attach populator
template void grid_test2<grid2r_attach_data>(grid2r_attach_data& grid_data);

// test2 function implementation
template <typename grid2_data_t>
void grid_test2(grid2_data_t& grid_data) {

    const auto& axis0 = grid_data._axis_p0;
    const auto& axis1 = grid_data._axis_p1;

    int num_blocks = axis0.bins() * axis1.bins();
    int num_threads = n_points;

    // run the kernel
    grid_test2_kernel<<<num_blocks, num_threads>>>(grid_data);

    // cuda error check
    DETRAY_CUDA_ERROR_CHECK(hipGetLastError());
    DETRAY_CUDA_ERROR_CHECK(hipDeviceSynchronize());
}

// test2 kernel implementation
template <typename grid_data_t>
__global__ void grid_test2_kernel(grid_data_t grid_data) {
    /*
    typename grid_data_t::populator_t::device_vector_t data_device(
        grid_data._data_serialized);
    const auto& axis0 = grid_data._axis_p0;
    const auto& axis1 = grid_data._axis_p1;

    auto& bin_data = data_device[blockIdx.x][threadIdx.x];
    auto& pt = bin_data;

    auto x_interval = (axis0.max - axis0.min) / axis0.n_bins;
    auto y_interval = (axis1.max - axis1.min) / axis1.n_bins;

    auto gid = threadIdx.x + blockIdx.x * blockDim.x;

    pt = test::point3{axis0.min + gid * x_interval,
                      axis1.min + gid * y_interval, 0.5};
    */
}

/*----------------------------------------------------
  test function for grid buffer with attach populator
  ----------------------------------------------------*/

// buffer_test kernel declaration
template <typename grid_data_t>
__global__ void grid_buffer_test_kernel(grid_data_t grid_data);

// buffer_test instantiation for attach populator

template void grid_buffer_test<grid2r_attach_data>(
    grid2r_attach_data& grid_data);

template <typename grid2_data_t>
void grid_buffer_test(grid2_data_t& grid_data) {

    const auto& axis0 = grid_data._axis_p0;
    const auto& axis1 = grid_data._axis_p1;

    dim3 block_dim(axis0.bins(), axis1.bins());
    int thread_dim = 1;

    // run the kernel
    grid_buffer_test_kernel<<<block_dim, thread_dim>>>(grid_data);

    // cuda error check
    DETRAY_CUDA_ERROR_CHECK(hipGetLastError());
    DETRAY_CUDA_ERROR_CHECK(hipDeviceSynchronize());
}

// buffer_test kernel declaration
template <typename grid_data_t>
__global__ void grid_buffer_test_kernel(grid_data_t grid_data) {

    /*
    using grid2_device_t = grid2<grid_data_t::populator_t,
                                 grid_data_t::axis_p0_t, grid_data_t::axis_po_1,
                                 grid_data_t::serializer_t>
    */
    // Let's try building the grid object
    grid2r_attach_device g2_device(grid_data, test::point3{0, 0, 0});

    // Fill with 10 points
    for (int i = 0; i < 10; i++) {
        auto pt = test::point3{0, 0, 0};
        g2_device.populate(blockIdx.x, blockIdx.y, std::move(pt));
    }
}

}  // namespace detray
