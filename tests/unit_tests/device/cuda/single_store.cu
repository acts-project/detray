#include "hip/hip_runtime.h"
/** Detray library, part of the ACTS project (R&D line)
 *
 * (c) 2022-2023 CERN for the benefit of the ACTS project
 *
 * Mozilla Public License Version 2.0
 */

// Project include(s).
#include <vecmem/memory/cuda/device_memory_resource.hpp>
#include <vecmem/memory/host_memory_resource.hpp>
#include <vecmem/utils/cuda/copy.hpp>

#include "detray/core/detail/single_store_test.hpp"

// GTest include(s).
#include <gtest/gtest.h>

#include <iostream>

using namespace detray;

using store_t = single_store<int>;
using store_dev_t = single_store<int, vecmem::device_vector>;

__global__ void basic_kernel(typename store_t::view_type view) {
    std::size_t globalIdx = threadIdx.x + blockIdx.x * blockDim.x;

    store_dev_t store(view);

    if (globalIdx >= store.size()) {
        return;
    }

    printf("%d\n", store.at(globalIdx, empty_context{}));
}

GTEST_TEST(detray_detail, single_store) {

    vecmem::host_memory_resource host_mr;
    store_t store(host_mr);
    store.push_back(3);
    store.push_back(4);

    vecmem::cuda::device_memory_resource dev_mr;
    vecmem::cuda::copy cpy;

    auto store_buffer = store.get_buffer(dev_mr, cpy);
    auto store_dev_view = detray::get_data(store_buffer);

    basic_kernel<<<8, 1>>>(store_dev_view);
}
