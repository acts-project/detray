#include "hip/hip_runtime.h"
/** Detray library, part of the ACTS project (R&D line)
 *
 * (c) 2022 CERN for the benefit of the ACTS project
 *
 * Mozilla Public License Version 2.0
 */

#include "detray/definitions/detail/cuda_definitions.hpp"
#include "navigator_cuda_kernel.hpp"

namespace detray {

__global__ void navigator_test_kernel(
    typename detector_host_t::view_type det_data, navigation::config cfg,
    vecmem::data::vector_view<free_track_parameters<algebra_t>> tracks_data,
    vecmem::data::jagged_vector_view<dindex> volume_records_data,
    vecmem::data::jagged_vector_view<point3> position_records_data) {

    int gid = threadIdx.x + blockIdx.x * blockDim.x;

    detector_device_t det(det_data);
    vecmem::device_vector<free_track_parameters<algebra_t>> tracks(tracks_data);
    vecmem::jagged_device_vector<dindex> volume_records(volume_records_data);
    vecmem::jagged_device_vector<point3> position_records(
        position_records_data);

    if (gid >= tracks.size()) {
        return;
    }

    navigator_device_t nav;

    auto& traj = tracks.at(gid);
    stepper_t stepper;

    prop_state<navigator_device_t::state> propagation{
        stepper_t::state{traj}, navigator_device_t::state(det)};

    navigator_device_t::state& navigation = propagation._navigation;
    stepper_t::state& stepping = propagation._stepping;

    // Set initial volume
    navigation.set_volume(0u);

    // Start propagation and record volume IDs
    bool heartbeat = nav.init(propagation, cfg);
    while (heartbeat) {

        heartbeat &= stepper.step(propagation);

        navigation.set_high_trust();

        heartbeat = nav.update(propagation, cfg);

        // Record volume
        volume_records[gid].push_back(navigation.volume());
        position_records[gid].push_back(stepping().pos());
    }
}

void navigator_test(
    typename detector_host_t::view_type det_data, navigation::config& cfg,
    vecmem::data::vector_view<free_track_parameters<algebra_t>>& tracks_data,
    vecmem::data::jagged_vector_view<dindex>& volume_records_data,
    vecmem::data::jagged_vector_view<point3>& position_records_data) {

    constexpr int thread_dim = 2 * WARP_SIZE;
    constexpr int block_dim = theta_steps * phi_steps / thread_dim + 1;

    // run the test kernel
    navigator_test_kernel<<<block_dim, thread_dim>>>(
        det_data, cfg, tracks_data, volume_records_data, position_records_data);

    // cuda error check
    DETRAY_CUDA_ERROR_CHECK(hipGetLastError());
    DETRAY_CUDA_ERROR_CHECK(hipDeviceSynchronize());
}

}  // namespace detray
